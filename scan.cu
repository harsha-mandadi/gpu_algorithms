#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void scan_inclusion(float * d_in,float * d_out){
	//index = threadIdx.x + (blockDim.x * blockIdx.x);
	int tid   = threadIdx.x;

	extern __shared__ float  sdata[10];

	sdata[tid] = d_in[tid];
	__syncthreads();

	float temp;
	for (int s=1;s<blockDim.x/2;s=s<<1) {
		
		if(tid>=s) {
			temp = sdata[tid]+sdata[tid-s];
			__syncthreads();
			sdata[tid] = temp;
		}	
		__syncthreads();
	}
       d_out[tid] = sdata[tid];	
}

int main(){
	const int  maxthreadsinablock = 10;
        const int THREADS= 10;
	const int BLOCKS = 1;
	const int ARRAY_SIZE = 10;

	//input array on the host
	float h_in[ARRAY_SIZE];
	float h_out[ARRAY_SIZE];

	//Generate input array for testing
	for (int i=0;i<ARRAY_SIZE;i++) {
		h_in[i] = float(i);
	}
	//declare GPU memory pointers
	float *d_in;
	float *d_out;

	//allocate memory on GPU
	hipMalloc((void **) &d_in,sizeof(float)*10);
	hipMalloc((void **) &d_out,sizeof(float)*10);

	hipMemcpy(d_in, h_in, sizeof(float)*10,hipMemcpyHostToDevice);

	scan_inclusion<<<BLOCKS,THREADS>>>(d_in,d_out);

	hipMemcpy(h_out, d_out, sizeof(float)*10,hipMemcpyDeviceToHost);


	//printing results
	for (int j=0;j<ARRAY_SIZE;j++) {
		printf ("%f\t",h_out[j]);
	}
	hipFree(d_in);
	hipFree(d_out);

return 0;
}

	
